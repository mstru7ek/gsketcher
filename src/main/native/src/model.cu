#include "hip/hip_runtime.h"
#include "model.cuh"

#include <stdio.h>
#include <stdlib.h>

namespace graph {

#ifdef __NVCC__
__GPU_DEV_INL__ Tensor<graph::BlockLayout> Vector::cartesian(Vector const &rhs) {
    double a00 = this->x * rhs.x;
    double a01 = this->x * rhs.y;
    double a10 = this->y * rhs.x;
    double a11 = this->y * rhs.y;
    return Tensor<graph::BlockLayout>(a00, a01, a10, a11);
}

__GPU_DEV_INL__ Vector Vector::Rot(double angle) {
    double rad = toRadians(angle);
    return Vector(this->x * cos(rad) - this->y * sin(rad), this->x * sin(rad) + this->y * cos(rad));
}

#endif

int constraintSize(Constraint const &constraint) {
    switch (constraint.constraintTypeId) {
    case CONSTRAINT_TYPE_ID_FIX_POINT:
        return 2;
    case CONSTRAINT_TYPE_ID_PARAMETRIZED_XFIX:
        return 1;
    case CONSTRAINT_TYPE_ID_PARAMETRIZED_YFIX:
        return 1;
    case CONSTRAINT_TYPE_ID_CONNECT_2_POINTS:
        return 2;
    case CONSTRAINT_TYPE_ID_HORIZONTAL_POINT:
        return 1;
    case CONSTRAINT_TYPE_ID_VERTICAL_POINT:
        return 1;
    case CONSTRAINT_TYPE_ID_LINES_PARALLELISM:
        return 1;
    case CONSTRAINT_TYPE_ID_LINES_PERPENDICULAR:
        return 1;
    case CONSTRAINT_TYPE_ID_EQUAL_LENGTH:
        return 1;
    case CONSTRAINT_TYPE_ID_PARAMETRIZED_LENGTH:
        return 1;
    case CONSTRAINT_TYPE_ID_TANGENCY:
        return 1;
    case CONSTRAINT_TYPE_ID_CIRCLE_TANGENCY:
        return 1;
    case CONSTRAINT_TYPE_ID_DISTANCE_2_POINTS:
        return 1;
    case CONSTRAINT_TYPE_ID_DISTANCE_POINT_LINE:
        return 1;
    case CONSTRAINT_TYPE_ID_ANGLE_2_LINES:
        return 1;
    case CONSTRAINT_TYPE_ID_SET_HORIZONTAL:
        return 1;
    case CONSTRAINT_TYPE_ID_SET_VERTICAL:
        return 1;

    default:
        printf("unknown constraint type \n");
        exit(1);
    }
}

int geometricSetSize(Geometric const &geometric) {
    switch (geometric.geometricTypeId) {
    case GEOMETRIC_TYPE_ID_FREE_POINT:
        return 3 * 2;
    case GEOMETRIC_TYPE_ID_LINE:
        return 4 * 2;
    case GEOMETRIC_TYPE_ID_CIRCLE:
        return 4 * 2;
    case GEOMETRIC_TYPE_ID_ARC:
        return 7 * 2;
    default:
        printf("unknown geometric type \n");
        exit(1);
    }
}

ComputationMode getComputationMode(int computationId) {
    switch (computationId) {
    case 1:
        return ComputationMode::DENSE_LAYOUT;
    case 2:
        return ComputationMode::SPARSE_LAYOUT;
    case 3:
        return ComputationMode::DIRECT_LAYOUT;
    default:
        printf("unknown computation id !\n");
        exit(1);
    }
}

/// accWriteCooStiff
///
/// __device__ __host__ COO tensor format requirments
///
int tensorOpsCooStiffnesCoefficients(Geometric const &geometric) {
    switch (geometric.geometricTypeId) {
    case GEOMETRIC_TYPE_ID_FREE_POINT:
        return 7 * 4; // 7 - plusSubTensor  * diagonal I (2)
    case GEOMETRIC_TYPE_ID_LINE:
        return 10 * 4; // 10 - plusSubTensor * diagonal I (2)
    case GEOMETRIC_TYPE_ID_CIRCLE:
        return 10 * 4; // 10 - plusSubTensor * diagonal I (2)
    case GEOMETRIC_TYPE_ID_ARC:
        return 19 * 4; // 19 - plusSubTensor * diagonal I (2)
    default:
        printf("unknown geometric type \n");
        exit(1);
    }
}
/// accWriteCooConstraint
int tensorOpsCooConstraintJacobian(Constraint const &constraint) {
    switch (constraint.constraintTypeId) {
    case CONSTRAINT_TYPE_ID_FIX_POINT:
        /// 1 * diagonal (4)
        return 4;
    case CONSTRAINT_TYPE_ID_PARAMETRIZED_XFIX:
        ///  1 * quick
        return 1;
    case CONSTRAINT_TYPE_ID_PARAMETRIZED_YFIX:
        ///  1 * quick
        return 1;
    case CONSTRAINT_TYPE_ID_CONNECT_2_POINTS:
        /// 2 * diagonal (4)
        return 8;
    case CONSTRAINT_TYPE_ID_HORIZONTAL_POINT:
        /// 2 * quick
        return 2;
    case CONSTRAINT_TYPE_ID_VERTICAL_POINT:
        ///  2 * quick
        return 2;
    case CONSTRAINT_TYPE_ID_LINES_PARALLELISM:
        /// 4 * vector(2)
        return 8;
    case CONSTRAINT_TYPE_ID_LINES_PERPENDICULAR:
        /// 4 * vector(2)
        return 8;
    case CONSTRAINT_TYPE_ID_EQUAL_LENGTH:
        /// 4 * vector(2)
        return 8;
    case CONSTRAINT_TYPE_ID_PARAMETRIZED_LENGTH:
        /// 4 * vector(2)
        return 8;
    case CONSTRAINT_TYPE_ID_TANGENCY:
        /// 4 * vector(2)
        return 8;
    case CONSTRAINT_TYPE_ID_CIRCLE_TANGENCY:
        /// 4 * vector(2)
        return 8;
    case CONSTRAINT_TYPE_ID_DISTANCE_2_POINTS:
        /// 2 * vector(2)
        return 4;
    case CONSTRAINT_TYPE_ID_DISTANCE_POINT_LINE:
        /// 3 * vector(2)
        return 6;
    case CONSTRAINT_TYPE_ID_ANGLE_2_LINES:
        /// 4 * vector(2)
        return 8;
    case CONSTRAINT_TYPE_ID_SET_HORIZONTAL:
        /// 2 * vector(2)
        return 4;
    case CONSTRAINT_TYPE_ID_SET_VERTICAL:
        /// 2 * vector(2)
        return 4;
    default:
        printf("unknown constraint type \n");
        exit(1);
    }
}

} // namespace graph