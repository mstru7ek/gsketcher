#include "hip/hip_runtime.h"
#include "tensor_operation.h"

#include "stdio.h"

TensorOperation::TensorOperation(hipStream_t stream) : stream(stream), pBuffer(NULL), Prm(NULL), Psize(0) {

    status = hipsparseCreate(&cuSparseHandle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        const char* errorName = hipsparseGetErrorName(status);
        const char* errorStr = hipsparseGetErrorString(status);
        fprintf(stderr, "[gpu/cusparse] cusparse initialization failure ; ( %s ) %s \n", errorName, errorStr);
        exit(1);
    }
    status = hipsparseSetStream(cuSparseHandle, stream);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        const char *errorName = hipsparseGetErrorName(status);
        const char *errorStr = hipsparseGetErrorString(status);
        fprintf(stderr, "[gpu/cusparse] cusparse stream failure ; ( %s ) %s \n", errorName, errorStr);
        exit(1);
    }
};

void TensorOperation::convertToCSR(int *cooRowInd, int *cooColInd, double *cooVal, int nnz, int ld, int *csrRowInd) {
           
    /// Ensure required buffer size
    int buffSize = pBufferSizeInBytes;

    status = hipsparseXcoosort_bufferSizeExt(cuSparseHandle, ld, ld, nnz, cooRowInd, cooColInd, &pBufferSizeInBytes);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        const char *errorName = hipsparseGetErrorName(status);
        const char *errorStr = hipsparseGetErrorString(status);
        fprintf(stderr, "[gpu/cusparse] cusparse stream failure ; ( %s ) %s \n", errorName, errorStr);
        exit(1);
    }

    if (pBufferSizeInBytes > buffSize) {

        if (pBuffer != NULL) {
            hipFreeAsync(pBuffer, stream);
        }
        /// Alllocate temporary buffer
        hipMallocAsync((void **)&pBuffer, pBufferSizeInBytes, stream);
    }

    if (nnz > Psize) {

        if (Prm != NULL) {

        }
    }



    // # PERMUTATION
    hipsparseCreateIdentityPermutation(cuSparseHandle, nnz, Prm);

    /// ONLY ONCE AS PRE-PROCESSING FOR SECOND ROUND

    // d_cooRowInd ?? - first round computed by memcpy !!!
    // d_cooColInd ?? - first round computed by memcpy !!!

    /// ===================================================== ///

    //# PROCEDURE SORT IN ROW  - destructive
    hipsparseXcoosortByRow(cuSparseHandle, ld, ld, nnz,
                          cooRowInd, // IN.OUT
                          cooColInd, // IN.OUT
                          Prm,             // IN.OUT
                          pBuffer);

    /// ===================================================== ///

    // # GATHER ELEMETNS
    hipsparseDgthr(cuSparseHandle, nnz,
                  cooVal, // IN
                  cooVal, // OUT
                  Prm, HIPSPARSE_INDEX_BASE_ZERO);

    /// Inverse all indicies for Direct Layout processing
    //inverse_indices(Prm, Pnvi, nnz);


    // hipsparseStatus_t hipsparseXcoo2csr(hipsparseHandle_t handle, const int *cooRowInd, int nnz, int m, int *csrRowPtr,hipsparseIndexBase_t idxBase);


}

TensorOperation::~TensorOperation() {
    //
    hipError_t hipError_t; 
    if (Prm != NULL) {
        hipError_t = hipFreeAsync(Prm, stream);
        if (hipError_t != hipSuccess) {
            const char *errorName = hipGetErrorName(hipError_t);
            const char *errorStr = hipGetErrorString(hipError_t);
            fprintf(stderr, "[gpu/cusparse] async free failed; ( %s ) %s \n", errorName, errorStr);
        }
    }

    if (pBuffer != NULL) {
        hipFreeAsync(pBuffer, stream);
        if (hipError_t != hipSuccess) {
            const char *errorName = hipGetErrorName(hipError_t);
            const char *errorStr = hipGetErrorString(hipError_t);
            fprintf(stderr, "[gpu/cusparse] async free failed; ( %s ) %s \n", errorName, errorStr);
        }
    }        

    if (cuSparseHandle != NULL) {

        status = hipsparseDestroy(cuSparseHandle);
        if (status != HIPSPARSE_STATUS_SUCCESS) {
            const char *errorName = hipsparseGetErrorName(status);
            const char *errorStr = hipsparseGetErrorString(status);
            fprintf(stderr, "[gpu/cusparse] handler destroy failed; ( %s ) %s \n", errorName, errorStr);
            exit(1);
        }
    }
}