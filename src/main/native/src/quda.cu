#include "hip/hip_runtime.h"
#include "quda.cuh"

#include <string>

#define KERNEL_EXECUTOR

namespace utility {




template <> struct printer<int> {
    __device__ __host__ void operator()(int i, int object) { 
        ///
        printf("%d  %d \n", i, object);
    }
};

template <> struct printer<double> {
    __device__ __host__ void operator()(int i, double object) {
        printf("%d  %f\n", i , object);
    }
};



template <typename Type> __global__ void __stdout_vector_kernel__(Type *vector, int size) {
    printer<Type> printer;
    int i = size;
    while (i-- > 0) {
        printer(i, vector[i]);
    }
}

/// <summary>
/// Device kernel executor for debug stdout from device.
/// </summary>
/// <typeparam name="Type"></typeparam>
/// <param name="vector"></param>
/// <param name="size"></param>

KERNEL_EXECUTOR template<typename Type> void stdout_vector_kernel(hipStream_t stream, Type *vector, int size) {
    //
    const unsigned GRID_DIM = 1;
    const unsigned BLOCK_DIM = 1;    
    const unsigned NS = 0;

    __stdout_vector_kernel__<Type><<<GRID_DIM, BLOCK_DIM, NS, stream>>>(vector, size);
}

template void stdout_vector_kernel<double>(hipStream_t stream, double *vector, int size);
template void stdout_vector_kernel<int>(hipStream_t stream, int *vector, int size);

} // namespace quda